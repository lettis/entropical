#include "hip/hip_runtime.h"

#include "densities_cuda.hpp"
#include "tools.hpp"

#include <hip/hip_runtime.h>

#define WGSIZE 128
#define N_STREAMS 4

namespace {

  //// CUDA kernels

  /* local probability from Epanechnikov kernel */
  __device__ float
  epa(float x
    , float ref_scaled_neg
    , float h_inv) {
    float p = fma(h_inv, x, ref_scaled_neg);
    p *= p;
    if (p <= 1.0f) {
      p = h_inv * fma(p, -0.75f, 0.75f);
    } else {
      p = 0.0f;
    }
    return p;
  }

  /* pre-reduce probabilities inside block */
  __device__ void
  local_prob_reduction(unsigned int tid
                     , float* p_wg
                     , float* P_partial) {
    unsigned int stride;
    unsigned int bid = blockIdx.x;
    // reduce locally
    for (stride=WGSIZE/2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        p_wg[tid] += p_wg[tid+stride];
      }
    }
    if (tid == 0) {
      P_partial[bid] = p_wg[0] / ((float) WGSIZE);
    }
  }

  /* compute partial 1D probabilities */
  __global__ void
  partial_probs_1d(unsigned int offset
                 , const float* sorted_coords
                 , unsigned int n_rows
                 , float* P_partial
                 , float h_inv_1
                 , float ref_scaled_neg_1) {
    __shared__ float p_wg[WGSIZE];
    unsigned int bid = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int gid = bid*bsize+tid + offset;
    // probability for every frame
    if (gid < n_rows) {
      p_wg[tid] = epa(sorted_coords[gid]
                    , ref_scaled_neg_1
                    , h_inv_1);
    } else {
      p_wg[tid] = 0.0f;
    }
    // pre-reduce inside workgroup
    local_prob_reduction(tid, p_wg, P_partial);
  }

  /* compute partial 2D probabilities */
  __global__ void
  partial_probs_2d(unsigned int offset
                 , const float* sorted_coords
                 , unsigned int n_rows
                 , float* P_partial
                 , float h_inv_1
                 , float ref_scaled_neg_1
                 , float h_inv_2
                 , float ref_scaled_neg_2) {
    __shared__ float p_wg[WGSIZE];
    unsigned int bid = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int gid = bid*bsize+tid + offset;
    // probability for every frame
    if (gid < n_rows) {
      p_wg[tid] = epa(sorted_coords[gid]
                    , ref_scaled_neg_1
                    , h_inv_1)
                * epa(sorted_coords[n_rows+gid]
                    , ref_scaled_neg_2
                    , h_inv_2);
    } else {
      p_wg[tid] = 0.0f;
    }
    // pre-reduce inside workgroup
    local_prob_reduction(tid, p_wg, P_partial);
  }

  /* compute partial 3D probabilities */
  __global__ void
  partial_probs_3d(unsigned int offset
                 , const float* sorted_coords
                 , unsigned int n_rows
                 , float* P_partial
                 , float h_inv_1
                 , float ref_scaled_neg_1
                 , float h_inv_2
                 , float ref_scaled_neg_2
                 , float h_inv_3
                 , float ref_scaled_neg_3) {
    __shared__ float p_wg[WGSIZE];
    unsigned int bid = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int gid = bid*bsize+tid + offset;
    // probability for every frame
    if (gid < n_rows) {
      p_wg[tid] = epa(sorted_coords[gid]
                    , ref_scaled_neg_1
                    , h_inv_1)
                * epa(sorted_coords[n_rows+gid]
                    , ref_scaled_neg_2
                    , h_inv_2)
                * epa(sorted_coords[2*n_rows+gid]
                    , ref_scaled_neg_3
                    , h_inv_3);
    } else {
      p_wg[tid] = 0.0f;
    }
    // pre-reduce inside workgroup
    local_prob_reduction(tid, p_wg, P_partial);
  }

  /* reduce partial probabilities */
  __global__ void
  sum_partial_probs_atomic(float* P_partial
                         , float* P
                         , unsigned int i_ref
                         , unsigned int n_partials
                         , unsigned int n_wg) {
    __shared__ float p_wg[WGSIZE];
    unsigned int stride;
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid*bsize+tid;
    unsigned int gid2 = gid + bsize*gridDim.x;
    // store probs locally for reduction
    if (gid2 < n_partials) {
      // initial double load and first reduction
      p_wg[tid] = P_partial[gid] + P_partial[gid2];
    } else if (gid < n_partials) {
      p_wg[tid] = P_partial[gid];
    } else {
      p_wg[tid] = 0.0f;
    }
    for (stride=WGSIZE/2; stride > 32; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        p_wg[tid] += p_wg[tid+stride];
      }
    }
    // unroll loop inside warp (intrinsic sync!)
    __syncthreads();
    if (tid < 32) {
      p_wg[tid] += p_wg[tid+32];
      p_wg[tid] += p_wg[tid+16];
      p_wg[tid] += p_wg[tid+8];
      p_wg[tid] += p_wg[tid+4];
      p_wg[tid] += p_wg[tid+2];
      p_wg[tid] += p_wg[tid+1];
    }
    if (tid == 0) {
      atomicAdd(&P[i_ref], p_wg[0] / ((float) n_wg));
    }
  }

} // end local namespace


std::vector<float>
combined_densities(const float* coords
                 , std::size_t n_rows
                 , std::vector<std::size_t> i_cols
                 , std::vector<float> h
                 , std::vector<std::size_t> tau) {
  std::size_t n_dim = i_cols.size();
  if (n_dim < 1 || 3 < n_dim) {
    std::cerr << "error: can only compute combined probabilities in 1, 2 or 3 "
              << "dimensions."
              << std::endl;
    exit(EXIT_FAILURE);
  }
  if (h.size() != n_dim) {
    std::cerr << "error: number of bandwidth parameters does not match number "
              << "of selected columns."
              << std::endl;
    exit(EXIT_FAILURE);
  }
  if (tau.size() != n_dim) {
    std::cerr << "error: number of lagtimes (tau) does not match number of "
              << "selected dimensions."
              << std::endl;
    exit(EXIT_FAILURE);
  }
  //TODO include lagtimes 'tau' in coord preparation and densities
  std::vector<float> sorted_coords = Tools::dim1_sorted_coords(coords
                                                             , n_rows
                                                             , i_cols
                                                             , tau);
  // create buffers on device
  unsigned int n_blocks = Tools::min_multiplicator(n_rows
                                                 , WGSIZE);
  unsigned int partial_size = Tools::min_multiplicator(n_blocks
                                                     , WGSIZE) * WGSIZE;
  float* d_sorted_coords;
  hipMalloc((void**) &d_sorted_coords
           , sizeof(float) * n_blocks*WGSIZE*n_dim);
  float* d_P;
  hipMalloc((void**) &d_P
           , sizeof(float) * n_rows);
  // one partial result array per stream
  float* d_P_partial[N_STREAMS];
  // copy coords to device
  hipMemcpy(d_sorted_coords
           , sorted_coords.data()
           , sizeof(float) * n_rows * n_dim
           , hipMemcpyHostToDevice);
  // initialize P with zeros
  hipMemset(d_P, 0, sizeof(float) * n_rows);
  // box limits for pruning
  std::vector<float> blimits = Tools::boxlimits(sorted_coords
                                              , WGSIZE
                                              , n_dim);
  // create multiple streams for parallel execution
  hipStream_t streams[N_STREAMS];
  for (unsigned int i=0; i < N_STREAMS; ++i) {
    hipStreamCreate(&streams[i]);
    hipMalloc((void**) &d_P_partial[i]
             , sizeof(float) * partial_size);
  }
  // run computation
  std::vector<float> h_inv(n_dim);
  for (std::size_t n=0; n < n_dim; ++n) {
    h_inv[n] = 1.0f/h[n];
  }
  for (unsigned int i=0; i < n_rows; ++i) {
    // set reference
    float ref_1 = coords[i_cols[0]*n_rows + i];
    std::vector<float> ref_scaled_neg(n_dim);
    for (std::size_t n=0; n < n_dim; ++n) {
      ref_scaled_neg[n] = -h_inv[n] * coords[i_cols[n]*n_rows + i];
    }
    // pruning on 1st dimension
    auto boxes_from_to = Tools::min_max_box(blimits
                                          , ref_1
                                          , h[0]);
    unsigned int bfrom = boxes_from_to.first;
    unsigned int bto = boxes_from_to.second;
    unsigned int n_blocks_pruned = bto - bfrom + 1;
    unsigned int offset = bfrom * WGSIZE;
    unsigned int rng = Tools::min_multiplicator(n_blocks_pruned
                                              , WGSIZE)
                     * WGSIZE;
    unsigned int i_stream = i % N_STREAMS;
    // compute partials asynchronously on different streams
    switch(n_dim) {
    case 1:
      partial_probs_1d <<< n_blocks_pruned
                         , WGSIZE
                         , 0
                         , streams[i_stream] >>> (offset
                                                , d_sorted_coords
                                                , n_rows
                                                , d_P_partial[i_stream]
                                                , h_inv[0]
                                                , ref_scaled_neg[0]);
      break;
    case 2:
      partial_probs_2d <<< n_blocks_pruned
                         , WGSIZE
                         , 0
                         , streams[i_stream] >>> (offset
                                                , d_sorted_coords
                                                , n_rows
                                                , d_P_partial[i_stream]
                                                , h_inv[0]
                                                , ref_scaled_neg[0]
                                                , h_inv[1]
                                                , ref_scaled_neg[1]);
      break;
    case 3:
      partial_probs_3d <<< n_blocks_pruned
                         , WGSIZE
                         , 0
                         , streams[i_stream] >>> (offset
                                                , d_sorted_coords
                                                , n_rows
                                                , d_P_partial[i_stream]
                                                , h_inv[0]
                                                , ref_scaled_neg[0]
                                                , h_inv[1]
                                                , ref_scaled_neg[1]
                                                , h_inv[2]
                                                , ref_scaled_neg[2]);
      break;
    default:
      // this should never happen
      exit(EXIT_FAILURE);
    }
    // compute P(i) from partials
    sum_partial_probs_atomic <<< rng/2
                               , WGSIZE
                               , 0
                               , streams[i_stream] >>> (d_P_partial[i_stream]
                                                      , d_P
                                                      , i
                                                      , n_blocks_pruned
                                                      , n_blocks);
  }
  // get results from GPU
  std::vector<float> P(n_rows);
  // sync over streams
  hipDeviceSynchronize();
  hipMemcpy(P.data()
           , d_P
           , sizeof(float) * n_rows
           , hipMemcpyDeviceToHost);
  // free the mallocs!
  hipFree(d_sorted_coords);
  hipFree(d_P);
  for (unsigned int i=0; i < N_STREAMS; ++i) {
    hipFree(d_P_partial);
  }
  return P;
}

