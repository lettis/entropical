#include "hip/hip_runtime.h"

#include "densities_cuda.hpp"
#include "tools.hpp"

#include <array>

#include <omp.h>
#include <hip/hip_runtime.h>

#define WGSIZE 128
#define N_STREAMS 4

namespace {

  //// CUDA kernels

  /* local probability from Epanechnikov kernel */
  __device__ float
  epa(float x
    , float ref_scaled_neg
    , float h_inv) {
    float p = fma(h_inv, x, ref_scaled_neg);
    p *= p;
    if (p <= 1.0f) {
      p = h_inv * fma(p, -0.75f, 0.75f);
    } else {
      p = 0.0f;
    }
    return p;
  }

  /* pre-reduce probabilities inside block */
  __device__ void
  local_prob_reduction(unsigned int tid
                     , float* p_wg
                     , float* P_partial) {
    unsigned int stride;
    unsigned int bid = blockIdx.x;
    // reduce locally
    for (stride=WGSIZE/2; stride > 0; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        p_wg[tid] += p_wg[tid+stride];
      }
    }
    if (tid == 0) {
      P_partial[bid] = p_wg[0] / ((float) WGSIZE);
    }
  }

  /* compute partial 2D probabilities */
  __global__ void
  partial_probs_2d(unsigned int offset
                 , const float* sorted_coords
                 , unsigned int n_rows
                 , float* P_partial
                 , float h_inv_1
                 , float ref_scaled_neg_1
                 , float h_inv_2
                 , float ref_scaled_neg_2) {
    __shared__ float p_wg[WGSIZE];
    unsigned int bid = blockIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int tid = threadIdx.x;
    unsigned int gid = bid*bsize+tid + offset;
    // probability for every frame
    if (gid < n_rows) {
      p_wg[tid] = epa(sorted_coords[gid]
                    , ref_scaled_neg_1
                    , h_inv_1)
                * epa(sorted_coords[n_rows+gid]
                    , ref_scaled_neg_2
                    , h_inv_2);
    } else {
      p_wg[tid] = 0.0f;
    }
    // pre-reduce inside workgroup
    local_prob_reduction(tid, p_wg, P_partial);
  }


  __inline__ __device__
  float warpReduceSum(float val) {
    for (unsigned int offset = warpSize/2; offset > 0; offset /= 2) {
      val += __shfl_down(val, offset);
    }
    return val;
  }


  __inline__ __device__
  float blockReduceSum(float val) {
    // Shared mem for 32 partial sums
    static __shared__ int shared[32];
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    // Each warp performs partial reduction
    val = warpReduceSum(val);
    // Write reduced value to shared memory
    if (lane==0) {
      shared[wid]=val;
    }
    // Wait for all partial reductions
    __syncthreads();
    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if (wid==0) {
      val = warpReduceSum(val); //Final reduce within first warp
    }
    return val;
  }

  

  __global__ void
  sum_partial_probs_atomic(float* P_partial
                         , float* P
                         , unsigned int i_ref
                         , unsigned int n_partials
                         , unsigned int n_wg) {
    __shared__ float p_wg[WGSIZE];
    unsigned int stride;
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bsize = blockDim.x;
    unsigned int gid = bid*bsize+tid;
    unsigned int gid2 = gid + bsize*gridDim.x;
    // store probs locally for reduction
    if (gid2 < n_partials) {
      // initial double load and first reduction
      p_wg[tid] = P_partial[gid] + P_partial[gid2];
    } else if (gid < n_partials) {
      p_wg[tid] = P_partial[gid];
    } else {
      p_wg[tid] = 0.0f;
    }
    for (stride=WGSIZE/2; stride > 32; stride /= 2) {
      __syncthreads();
      if (tid < stride) {
        p_wg[tid] += p_wg[tid+stride];
      }
    }
    // unroll loop inside warp (intrinsic sync!)
    __syncthreads();
    if (tid < 32) {
      p_wg[tid] += p_wg[tid+32];
      p_wg[tid] += p_wg[tid+16];
      p_wg[tid] += p_wg[tid+8];
      p_wg[tid] += p_wg[tid+4];
      p_wg[tid] += p_wg[tid+2];
      p_wg[tid] += p_wg[tid+1];
    }
    if (tid == 0) {
      atomicAdd(&P[i_ref], p_wg[0] / ((float) n_wg));
    }
  }

  ////


  std::vector<float>
  densities_1d(const float* coords
             , std::vector<std::size_t> i_col
             , const std::vector<float>& sorted_coords
             , std::vector<float> h) {
    //TODO
    return {};
  }


  std::vector<float>
  densities_2d(const float* coords
             , std::vector<std::size_t> i_cols
             , const std::vector<float>& sorted_coords
             , std::vector<float> h) {
    std::size_t n_rows = sorted_coords.size() / 2;
    // create buffers on device
    unsigned int n_blocks = Tools::min_multiplicator(n_rows
                                                   , WGSIZE);
    unsigned int partial_size = Tools::min_multiplicator(n_blocks
                                                       , WGSIZE) * WGSIZE;
    float* d_sorted_coords;
    hipMalloc((void**) &d_sorted_coords
             , sizeof(float) * n_blocks*WGSIZE*2);
    float* d_P;
    hipMalloc((void**) &d_P
             , sizeof(float) * n_rows);
    // one partial result array per stream
    float* d_P_partial[N_STREAMS];
    // copy coords to device
    hipMemcpy(d_sorted_coords
             , sorted_coords.data()
             , sizeof(float) * n_rows * 2
             , hipMemcpyHostToDevice);
    // initialize P with zeros
    hipMemset(d_P, 0, sizeof(float) * n_rows);
    // run computation
    float h_inv_1 = 1/h[0];
    float h_inv_2 = 1/h[1];
    // box limits for pruning
    std::vector<float> blimits = Tools::boxlimits(sorted_coords
                                                , WGSIZE
                                                , 2);
    // create multiple streams for parallel execution
    hipStream_t streams[N_STREAMS];
    for (unsigned int i=0; i < N_STREAMS; ++i) {
      hipStreamCreate(&streams[i]);
      hipMalloc((void**) &d_P_partial[i]
               , sizeof(float) * partial_size);
    }
    for (unsigned int i=0; i < n_rows; ++i) {
      // set reference
      float ref_1 = coords[i_cols[0]*n_rows + i];
      float ref_scaled_neg_1 = -h_inv_1 * ref_1;
      float ref_scaled_neg_2 = -h_inv_2 * coords[i_cols[1]*n_rows + i];
      // pruning
      auto boxes_from_to = Tools::min_max_box(blimits
                                            , ref_1
                                            , h[0]);
      unsigned int bfrom = boxes_from_to.first;
      unsigned int bto = boxes_from_to.second;
      unsigned int n_blocks_pruned = bto - bfrom + 1;
      unsigned int offset = bfrom * WGSIZE;
      unsigned int rng = Tools::min_multiplicator(n_blocks_pruned
                                                , WGSIZE)
                       * WGSIZE;
      unsigned int i_stream = i % N_STREAMS;
      // compute partials asynchronously on different streams
      partial_probs_2d <<< n_blocks_pruned
                         , WGSIZE
                         , 0
                         , streams[i_stream] >>> (offset
                                                , d_sorted_coords
                                                , n_rows
                                                , d_P_partial[i_stream]
                                                , h_inv_1
                                                , ref_scaled_neg_1
                                                , h_inv_2
                                                , ref_scaled_neg_2);
      // compute P(i) from partials
      sum_partial_probs_atomic <<< rng/2
                                 , WGSIZE
                                 , 0
                                 , streams[i_stream] >>> (d_P_partial[i_stream]
                                                        , d_P
                                                        , i
                                                        , n_blocks_pruned
                                                        , n_blocks);
    }
    // get results from GPU
    std::vector<float> P(n_rows);
    // sync over streams
    hipDeviceSynchronize();
    hipMemcpy(P.data()
             , d_P
             , sizeof(float) * n_rows
             , hipMemcpyDeviceToHost);
    // free the mallocs!
    hipFree(d_sorted_coords);
    hipFree(d_P);
    for (unsigned int i=0; i < N_STREAMS; ++i) {
      hipFree(d_P_partial);
    }
    return P;
  }


  std::vector<float>
  densities_3d(const float* coords
             , std::vector<std::size_t> i_col
             , const std::vector<float>& sorted_coords
             , std::vector<float> h) {
    //TODO
    return {};
  }

} // end local namespace


std::vector<float>
combined_densities(const float* coords
                 , std::size_t n_rows
                 , std::vector<std::size_t> i_cols
                 , std::vector<float> h) {
  std::size_t n_dim = i_cols.size();
  std::vector<float> sorted_coords = Tools::dim1_sorted_coords(coords
                                                             , n_rows
                                                             , i_cols);
  switch(n_dim) {
    case 1:
      return densities_1d(coords
                        , i_cols
                        , sorted_coords
                        , h);
    case 2:
      return densities_2d(coords
                        , i_cols
                        , sorted_coords
                        , h);
    case 3:
      return densities_3d(coords
                        , i_cols
                        , sorted_coords
                        , h);
    default:
      // this should never happen!
      exit(EXIT_FAILURE);
  }
}
